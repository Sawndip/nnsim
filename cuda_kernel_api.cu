#include "hip/hip_runtime.h"
/*
 * cuda_kernel_api.cpp
 *
 *  Created on: 01 нояб. 2014 г.
 *      Author: pavel
 */

#include "cuda_kernel_declarations.h"
#include "nnsim_constants.h"

__device__ float get_random(unsigned int *seed){
	// return random number homogeneously distributed in interval [0:1]
	unsigned long a = 16807;
	unsigned long m = 2147483647;
	unsigned long x = (unsigned long) *seed;
	x = (a * x) % m;
	*seed = (unsigned int) x;
	return ((float)x)/m;
}

__global__ void integrate_synapses(float* x, float* u, float* exp_rec, float* exp_fac, float* U, 
								   float* weight, int* delay, int* pre_syn, int* post_syn, unsigned int* receptor_type,
								   unsigned int* syn_num_spk, unsigned int* neur_num_spk, unsigned int* spk_time,
								   float* AMPA_Amuont, float* GABA_Amuont,
								   unsigned int t, int Ncon, int Nneur){
	unsigned int c = blockDim.x*blockIdx.x + threadIdx.x;
	if (c < Ncon){
		x[c] = (x[c] - weight[c])*exp_rec[c] + weight[c];
		u[c] = u[c]*exp_fac[c];
	
		if (syn_num_spk[c] < neur_num_spk[pre_syn[c]]){
			if (t >= delay[c] && spk_time[Nneur*syn_num_spk[c] + pre_syn[c]] == t - delay[c]){
//				printf("Spike! neur: %i time: %f\n", post_syns[c], t*time_step);
				u[c] += U[c]*(1.0f - u[c]);
				float delta_x = x[c]*u[c];
				x[c] -= delta_x;
				syn_num_spk[c]++;
				
				// When run parallel this incrementation should be atomic
				if (receptor_type[c] == AMPA_RECEPTOR){
					atomicAdd(&AMPA_Amuont[post_syn[c]], delta_x);
				} else if (receptor_type[c] == GABA_RECEPTOR){
					atomicAdd(&GABA_Amuont[post_syn[c]], delta_x);
				}
			}
		}
	
	}
}

__device__ __inline__ float check_pow(float x, float degr){
	if (degr == 1.0f){
		return x;
	} else {
		return powf(x, degr);
	}
}

__global__ void integrate_neurons(float* Vms, float* Ums,
		float* a, float* b1, float* b2, float* c, float* d, float* k, float* p1, float* p2,
		float* Vpeak, float* Vr, float* Vt, float* Cm, float* Ie, float* Isyn,
		float* AMPA_Amuont, float* GABA_Amuont, float* exp_pscs_exc, float* exp_pscs_inh,
		float* Erev_exc, float* Erev_inh, 
		float* y_psn, float* psn_weight, float* exp_psn, unsigned int* psn_time, float* psn_rate, unsigned int* psn_seed,
		unsigned int* spk_time, unsigned int* neur_num_spk, 
		unsigned int t, float time_step, int Nneur){
	unsigned int n = blockIdx.x*blockDim.x + threadIdx.x;
	float v1, u1, v2, u2, v3, u3, v4, u4;
	if (n < Nneur){
		y_psn[n] *= exp_psn[n];
		while (psn_time[n] == t){
			y_psn[n] += psn_weight[n];
			psn_time[n] -= -1 + (1000.0f/(time_step*psn_rate[n]))*log(get_random(psn_seed + n));
		}

		AMPA_Amuont[n] *= exp_pscs_exc[n];
		GABA_Amuont[n] *= exp_pscs_inh[n];
	
		float Vm = Vms[n];
		float Um = Ums[n];
		// y_psns is here because poisson noise is excitatory
		float Isyn_new = -(AMPA_Amuont[n] + y_psn[n])*(Vm - Erev_exc[n]) - GABA_Amuont[n]*(Vm - Erev_inh[n]);

		v1 = (k[n]*(Vm - Vr[n])*(Vm - Vt[n]) - Um + Ie[n] + Isyn[n])*time_step/Cm[n];
		u1 = time_step*a[n]*(Vms[n] < Vr[n] ? b1[n]*check_pow((Vm - Vr[n]), p1[n]) - Um : b2[n]*check_pow((Vm - Vr[n]), p2[n]) - Um);
		Vms[n] = Vm + v1*0.5f;
		Ums[n] = Um + u1*0.5f;
		v2 = (k[n]*(Vms[n] - Vr[n])*(Vms[n] - Vt[n]) - Ums[n] + Ie[n] + (Isyn_new + Isyn[n])*0.5f)*time_step/Cm[n];
		u2 = time_step*a[n]*(Vms[n] < Vr[n] ? b1[n]*check_pow((Vms[n] - Vr[n]), p1[n]) - Ums[n] : b2[n]*check_pow((Vms[n] - Vr[n]), p2[n]) - Ums[n]);
		Vms[n] = Vm + v2*0.5f;
		Ums[n] = Um + u2*0.5f;
		v3 = (k[n]*(Vms[n] - Vr[n])*(Vms[n] - Vt[n]) - Ums[n] + Ie[n] + (Isyn_new + Isyn[n])*0.5f)*time_step/Cm[n];
		u3 = time_step*a[n]*(Vms[n] < Vr[n] ? b1[n]*check_pow((Vms[n] - Vr[n]), p1[n]) - Ums[n] : b2[n]*check_pow((Vms[n] - Vr[n]), p2[n]) - Ums[n]);
		Vms[n] = Vm + v3;
		Ums[n] = Um + u3;
		v4 = (k[n]*(Vms[n] - Vr[n])*(Vms[n] - Vt[n]) - Ums[n] + Ie[n] + Isyn_new)*time_step/Cm[n];
		u4 = time_step*a[n]*(Vms[n] < Vr[n] ? b1[n]*check_pow((Vms[n] - Vr[n]), p1[n]) - Ums[n] : b2[n]*check_pow((Vms[n] - Vr[n]), p2[n]) - Ums[n]);
		Vms[n] = Vm + (v1 + 2.0f*(v2 + v3) + v4)*0.16666666f;
		Ums[n] = Um + (u1 + 2.0f*(u2 + u3) + u4)*0.16666666f;

		if (Vm > Vpeak[n]){
//			printf("Spike! neur: %i time: %f\n", n, t*time_step);
			spk_time[Nneur*neur_num_spk[n] + n] = t;
			neur_num_spk[n]++;
			Vms[n] = c[n];
			Ums[n] = Um + d[n];
		}
		Isyn[n] = Isyn_new;
	}
}

void simulateOnGpu(){
	init_mem();
	copy2device();
	for (unsigned int t = 0; t < Tsim; t++){
		integrate_neurons<<<Nneur/NEUR_BLOCK_SZ + 1, NEUR_BLOCK_SZ>>>(
				Vms_dev, Ums_dev, as_dev, b1_s_dev, b2_s_dev, cs_dev, ds_dev, ks_dev, p1_s_dev, p2_s_dev,
				Vpeaks_dev, Vrs_dev, Vts_dev, Cms_dev, Ies_dev, Isyns_dev,
				AMPA_Amuont_dev, GABA_Amuont_dev, exp_pscs_exc_dev, exp_pscs_inh_dev,
				Erev_exc_dev, Erev_inh_dev,
				y_psns_dev, psn_weights_dev, exp_psns_dev, psn_times_dev, psn_rates_dev, psn_seeds_dev,
				spk_times_dev, neur_num_spks_dev, t, time_step, Nneur);
//		hipDeviceSynchronize();
//		integrate_synapses<<<Ncon/SYN_BLOCK_SZ + 1, SYN_BLOCK_SZ>>>(
//				);
//		hipDeviceSynchronize();
	}
	const char* error = hipGetErrorString(hipPeekAtLastError());
	printf("%s\n", error);
	error = hipGetErrorString(hipDeviceSynchronize());
	printf("%s\n", error);
	CUDA_CHECK_RETURN(
		hipMemcpy(spk_times, spk_times_dev, sizeof(unsigned int)*len_spk_tms, hipMemcpyDeviceToHost));
	CUDA_CHECK_RETURN(
		hipMemcpy(neur_num_spks, neur_num_spks_dev, sizeof(unsigned int)*Nneur, hipMemcpyDeviceToHost));
}
