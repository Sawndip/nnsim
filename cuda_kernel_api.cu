#include "hip/hip_runtime.h"
/*
 * cuda_kernel_api.cpp
 *
 *  Created on: 01 нояб. 2014 г.
 *      Author: pavel
 */

__constant__ unsigned int AMPA_RECEPTOR = 1;
__constant__ unsigned int GABA_RECEPTOR = 2;

__device__ float get_random(unsigned int *seed){
	// return random number homogeneously distributed in interval [0:1]
	unsigned long a = 16807;
	unsigned long m = 2147483647;
	unsigned long x = (unsigned long) *seed;
	x = (a * x) % m;
	*seed = (unsigned int) x;
	return ((float)x)/m;
}

__global__ void integrate_synapses(float* x, float* u, float* exp_rec, float* exp_fac, float* U, 
								   float* weight, int* delay, int* pre_syn, int* post_syn, unsigned int* receptor_type,
								   unsigned int* syn_num_spk, unsigned int* neur_num_spk, unsigned int* spk_time,
								   float* AMPA_Amuont, float* GABA_Amuont,
								   unsigned int t, int Ncon, int Nneur){
	unsigned int c = blockDim.x*blockIdx.x + threadIdx.x;
	if (c < Ncon){
		x[c] = (x[c] - weight[c])*exp_rec[c] + weight[c];
		u[c] = u[c]*exp_fac[c];
	
		if (syn_num_spk[c] < neur_num_spk[pre_syn[c]]){
			if (t >= delay[c] && spk_time[Nneur*syn_num_spk[c] + pre_syn[c]] == t - delay[c]){
//				printf("Spike! neur: %i time: %f\n", post_syns[c], t*time_step);
				u[c] += U[c]*(1.0f - u[c]);
				float delta_x = x[c]*u[c];
				x[c] -= delta_x;
				syn_num_spk[c]++;
				
				// When run parallel this incrementation should be atomic
				if (receptor_type[c] == AMPA_RECEPTOR){
					atomicAdd(&AMPA_Amuont[post_syn[c]], delta_x);
				} else if (receptor_type[c] == GABA_RECEPTOR){
					atomicAdd(&GABA_Amuont[post_syn[c]], delta_x);
				}
			}
		}
	
	}
}

__global__ void integrate_neurons(float* Vms, float* Ums,
		float* a, float* b1, float* b2, float* c, float* d, float* k, float* p1, float* p2,
		float* Vpeak, float* Vr, float* Vt, float* Cm, float* Ie, float* Isyn,
		float* AMPA_Amuont, float* GABA_Amuont, float* exp_pscs_exc, float* exp_pscs_inh,
		float* Erev_exc, float* Erev_inh, 
		float* y_psn, float* psn_weight, float* exp_psn, float* psn_time, float* psn_rate, unsigned int* psn_seed,
		unsigned int* spk_time, unsigned int* neur_num_spk, 
		int t, float time_step, int Nneur){
	unsigned int n = blockIdx.x*blockDim.x + threadIdx.x;
	float v1, u1, v2, u2, v3, u3, v4, u4;
	if (n < Neur){
		y_psn[n] *= exp_psn[n];
		while (psn_time[n] == t){
			y_psn[n] += psn_weight[n];
			psn_time[n] -= (1000.0f/(time_step*psn_rate[n]))*log(get_random(psn_seed + n));
		}
		AMPA_Amuont[n] *= exp_pscs_exc[n];
		GABA_Amuont[n] *= exp_pscs_inh[n];
	
		float Vm = Vms[n];
		float Um = Ums[n];
		// y_psns is here because poisson noise is excitatory
		float Isyn_new = -(AMPA_Amuont[n] + y_psn[n])*(Vm - Erev_exc[n]) - GABA_Amuont[n]*(Vm - Erev_inh[n]);

		v1 = (k[n]*(Vm - Vr[n])*(Vm - Vt[n]) - Um + Ie[n] + Isyn[n])*time_step/Cm[n];
		u1 = time_step*a[n]*(Vm < Vr[n] ? b1[n]*powf((Vm - Vr[n]), p1[n]) - Um : b2[n]*powf((Vm - Vr[n]), p2[n]) - Um);
		Vms[n] = Vm + v1*0.5f;
		Ums[n] = Um + u1*0.5f;
		v2 = (k[n]*(Vms[n] - Vr[n])*(Vms[n] - Vt[n]) - Ums[n] + Ie[n] + (Isyn_new + Isyn[n])*0.5f)*time_step/Cm[n];
		u2 = time_step*a[n]*(Vms[n] < Vr[n] ? b1[n]*powf((Vms[n] - Vr[n]), p1[n]) - Ums[n] : b2[n]*powf((Vms[n] - Vr[n]), p2[n]) - Ums[n]);
		Vms[n] = Vm + v2*0.5f;
		Ums[n] = Um + u2*0.5f;
		v3 = (k[n]*(Vms[n] - Vr[n])*(Vms[n] - Vt[n]) - Ums[n] + Ie[n] + (Isyn_new + Isyn[n])*0.5f)*time_step/Cm[n];
		u3 = time_step*a[n]*(Vms[n] < Vr[n] ? b1[n]*powf((Vms[n] - Vr[n]), p1[n]) - Ums[n] : b2[n]*powf((Vms[n] - Vr[n]), p2[n]) - Ums[n]);
		Vms[n] = Vm + v3;
		Ums[n] = Um + u3;
		v4 = (k[n]*(Vms[n] - Vr[n])*(Vms[n] - Vt[n]) - Ums[n] + Ie[n] + Isyn_new)*time_step/Cm[n];
		u4 = time_step*a[n]*(Vms[n] < Vr[n] ? b1[n]*powf((Vms[n] - Vr[n]), p1[n]) - Ums[n] : b2[n]*powf((Vms[n] - Vr[n]), p2[n]) - Ums[n]);
		Vms[n] = Vm + (v1 + 2.0f*(v2 + v3) + v4)*0.16666666f;
		Ums[n] = Um + (u1 + 2.0f*(u2 + u3) + u4)*0.16666666f;
	}
	
	if (Vm > Vpeak[n]){
//				printf("Spike! neur: %i time: %f\n", n, t*time_step);
		spk_time[Nneur*neur_num_spk[n] + n] = t;
		neur_num_spk[n]++;
		Vms[n] = c[n];
		Ums[n] = Um + d[n];
	}
	
	Isyn[n] = Isyn_new;
}
